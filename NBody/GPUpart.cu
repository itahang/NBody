#include "hip/hip_runtime.h"
﻿#include "GPUpart.cuh"
#include <hip/hip_runtime.h>

__device__ float2 normalizedVector(float2 vec, float epslon = 0.0001) {
	float dist_sq = vec.x * vec.x + vec.y * vec.y + epslon;
	float inv_dist = rsqrtf(dist_sq);
	return { vec.x * inv_dist, vec.y * inv_dist }; 
}

__device__ float2 calcAcceleration(Body* s, Body* t, float epsilon, float rfactor = 100.0) {
	float2 r = { rfactor * (t->position.x - s->position.x),rfactor * (t->position.y - s->position.y) };

	float rfac = powf(r.x * r.x + r.y * r.y, -1.5);
	float mass = s->mass * t->mass;

	return normalizedVector({ mass * rfac * r.x,mass * rfac * r.y });
}



__global__ void kernel(Body* d_pixels, int Width, int Height) {
	

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int idy = threadIdx.y + blockDim.y * blockIdx.y;

	const float dt = 0.001;
	const float epsilon = 1e-2;

	if (idx < Width && idy < Height) {
		int pixelIndex = idx + Width * idy;

		float2 old_position = d_pixels[pixelIndex].position;

		d_pixels[pixelIndex].acceleration = { 0, 0 };

		for (int i = 0; i < Width * Height; i++) {
			if (i != pixelIndex) {
				float2 tAcc = calcAcceleration(&d_pixels[pixelIndex], &d_pixels[i], epsilon);
				d_pixels[pixelIndex].acceleration.x += tAcc.x;
				d_pixels[pixelIndex].acceleration.y += tAcc.y;
			}
		}

		float2 new_position;
		new_position.x = 2 * old_position.x - d_pixels[pixelIndex].prev_position.x +
			dt * dt * d_pixels[pixelIndex].acceleration.x;
		new_position.y = 2 * old_position.y - d_pixels[pixelIndex].prev_position.y +
			dt * dt * d_pixels[pixelIndex].acceleration.y;

		d_pixels[pixelIndex].prev_position = old_position;
		d_pixels[pixelIndex].position = new_position;

	}
}

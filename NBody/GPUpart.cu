#include "hip/hip_runtime.h"
﻿#include "GPUpart.cuh"
#include <hip/hip_runtime.h>

__device__ double2 normalizedVector(double2 vec, double epslon = 0.0001) {
	double dist_sq = vec.x * vec.x + vec.y * vec.y + epslon;
	double inv_dist = rsqrt(dist_sq);
	return { vec.x * inv_dist, vec.y * inv_dist };
}

__device__ double2 calcAcceleration(Body* s, Body* t, double epsilon, double rfactor = 100.0) {
	double2 r = { rfactor * (t->position.x - s->position.x),rfactor * (t->position.y - s->position.y) };

	double rfac = pow(r.x * r.x + r.y * r.y, -1.5);
	double mass = s->mass * t->mass;

	return normalizedVector({ mass * rfac * r.x,mass * rfac * r.y });
}


extern __device__ double2 meanposition;

__global__ void kernel(Body* d_pixels, int Width, int Height) {


	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int idy = threadIdx.y + blockDim.y * blockIdx.y;

	const double dt = 0.001;
	const double epsilon = 1e-2;

	if (idx < Width && idy < Height) {
		int pixelIndex = idx + Width * idy;

		meanposition = d_pixels[0].position;
		__syncthreads();

		double2 old_position = d_pixels[pixelIndex].position;

		d_pixels[pixelIndex].acceleration = { 0, 0 };

		for (int i = 0; i < Width * Height; i++) {
			if (i != pixelIndex) {
				double2 tAcc = calcAcceleration(&d_pixels[pixelIndex], &d_pixels[i], epsilon);
				d_pixels[pixelIndex].acceleration.x += tAcc.x;
				d_pixels[pixelIndex].acceleration.y += tAcc.y;
			}
		}

		double2 new_position;
		new_position.x = 2 * old_position.x - d_pixels[pixelIndex].prev_position.x +
			dt * dt * d_pixels[pixelIndex].acceleration.x;
		new_position.y = 2 * old_position.y - d_pixels[pixelIndex].prev_position.y +
			dt * dt * d_pixels[pixelIndex].acceleration.y;

		d_pixels[pixelIndex].prev_position = old_position;
		d_pixels[pixelIndex].position = new_position;

		d_pixels[pixelIndex].position.x = d_pixels[pixelIndex].position.x - meanposition.x;
		d_pixels[pixelIndex].position.y = d_pixels[pixelIndex].position.y - meanposition.y;

	}

}